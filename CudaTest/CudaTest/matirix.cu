#include "hip/hip_runtime.h"
#include "matrix.cuh"

__global__ void MatrixMultiplicationKernel(int* M_d, int* N_d, int* P_d,
                                           int width) {
  // 2D Thread ID
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int result = 0;
  int M_dElement = 0;
  int N_dElement = 0;

  for (int k = 0; k < width; ++k) {
    M_dElement = M_d[ty * width + k];
    N_dElement = N_d[k * width + tx];
    result += M_dElement * N_dElement;
  }

  P_d[ty * width + tx] = result;
}

int MatrixMultiplication(int* M, int* N, int* P, int width) {
  int size = width * width * sizeof(int);
  int* M_d;
  int* N_d;
  int* P_d;

  // Transfer M and N to device memory
  hipError_t mallocStatus = hipMalloc((void**)&M_d, size);
  if (mallocStatus != hipSuccess) {
    std::cerr << "hipMalloc failed: " << hipGetErrorString(mallocStatus)
              << std::endl;
    return 1;
  }

  hipError_t memcpyStatus = hipMemcpy(M_d, M, size, hipMemcpyHostToDevice);
  if (memcpyStatus != hipSuccess) {
    std::cerr << "hipMemcpy failed: " << hipGetErrorString(memcpyStatus)
              << std::endl;
    hipFree(M_d);
    return 1;
  }

  mallocStatus = hipMalloc((void**)&N_d, size);
  if (mallocStatus != hipSuccess) {
    std::cerr << "hipMalloc failed: " << hipGetErrorString(mallocStatus)
              << std::endl;
    return 1;
  }

  memcpyStatus = hipMemcpy(N_d, N, size, hipMemcpyHostToDevice);
  if (memcpyStatus != hipSuccess) {
    std::cerr << "hipMemcpy failed: " << hipGetErrorString(memcpyStatus)
              << std::endl;
    hipFree(M_d);
    hipFree(N_d);
    return 1;
  }
  // Allocate P on the device
  mallocStatus = hipMalloc((void**)&P_d, size);
  if (mallocStatus != hipSuccess) {
    std::cerr << "hipMalloc failed: " << hipGetErrorString(mallocStatus)
              << std::endl;
    return 1;
  }

  // Ensure all preceding operations are complete
  hipError_t syncStatus = hipDeviceSynchronize();
  if (syncStatus != hipSuccess) {
    std::cerr << "hipDeviceSynchronize failed: "
              << hipGetErrorString(syncStatus) << std::endl;
    hipFree(M_d);
    hipFree(N_d);
    return 1;
  }

  // Setup the execution configuration
  dim3 dimBlock(width, width);
  dim3 dimGrid(1, 1);

  // Launch the device computation threads
  MatrixMultiplicationKernel<<<dimGrid, dimBlock>>>(M_d, N_d, P_d, width);

  // Transfer P from the device to the host
  memcpyStatus = hipMemcpy(P, P_d, size, hipMemcpyDeviceToHost);
  if (memcpyStatus != hipSuccess) {
    std::cerr << "hipMemcpy failed: " << hipGetErrorString(memcpyStatus)
              << std::endl;
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    return 1;
  }
  syncStatus = hipDeviceSynchronize();
  if (syncStatus != hipSuccess) {
    std::cerr << "hipDeviceSynchronize failed: "
              << hipGetErrorString(syncStatus) << std::endl;
    hipFree(M_d);
    hipFree(N_d);
    hipFree(P_d);
    return 1;
  }
  // Free device matrices
  hipFree(M_d);
  hipFree(N_d);
  hipFree(P_d);

  return 0;
}